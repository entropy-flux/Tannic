#include "hip/hip_runtime.h"
#include "cuda/cuda.cuh"
#include "cuda/unary-ops.cuh"
#include <hip/hip_runtime.h>
#include <cassert>

template<typename S, typename D, typename Op>
__global__ void unaryOpKernel(
    const S* __restrict__ src_data,
    D* __restrict__ dst_data,
    size_t total,
    uint8_t rank,
    const size_t* __restrict__ shape,
    const size_t* __restrict__ strides_src,
    size_t offset_src,
    size_t offset_dst,
    Op op
) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t counters[8] = {0};  // assume max rank is 8
    size_t residual = idx;
    for (int i = rank - 1; i >= 0; --i) {
        counters[i] = residual % shape[i];
        residual /= shape[i];
    }

    size_t src_offset = 0;
    for (int i = 0; i < rank; ++i) {
        src_offset += counters[i] * strides_src[i];
    }

    dst_data[offset_dst + idx] = op(src_data[offset_src + src_offset]);
}

template<typename S, typename D, typename Op>
void cuda::unaryOp(const tensor_t* src, tensor_t* dst, Op op, hipStream_t stream) {
    const auto rank = dst->rank;
    assert(rank <= 8 && "Rank exceeds kernel limit");

    const size_t* shape = dst->shape;
    const size_t* strides_src = src->strides;

    size_t total = 1;
    for (uint8_t i = 0; i < rank; ++i) {
        total *= shape[i];
    }

    const int threadsPerBlock = 256;
    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;

    unaryOpKernel<S, D, Op><<<blocks, threadsPerBlock, 0, stream>>>(
        static_cast<const S*>(src->storage->address),
        static_cast<D*>(dst->storage->address),
        total,
        rank,
        shape,
        strides_src,
        src->offset,
        dst->offset,
        op
    );
    CUDA_CHECK(hipGetLastError());

#ifndef NDEBUG
    CUDA_CHECK(hipStreamSynchronize(stream));
#endif
}


// TODO: Explicit template instatiation should be refactor with macros. 
// NEGATION
template void cuda::unaryOp<int8_t, int8_t, cuda::negation_op::Negation>(const tensor_t*, tensor_t*, cuda::negation_op::Negation, hipStream_t);
template void cuda::unaryOp<int16_t, int16_t, cuda::negation_op::Negation>(const tensor_t*, tensor_t*, cuda::negation_op::Negation, hipStream_t);
template void cuda::unaryOp<int32_t, int32_t, cuda::negation_op::Negation>(const tensor_t*, tensor_t*, cuda::negation_op::Negation, hipStream_t);
template void cuda::unaryOp<int64_t, int64_t, cuda::negation_op::Negation>(const tensor_t*, tensor_t*, cuda::negation_op::Negation, hipStream_t);
template void cuda::unaryOp<float, float, cuda::negation_op::Negation>(const tensor_t*, tensor_t*, cuda::negation_op::Negation, hipStream_t);
template void cuda::unaryOp<double, double, cuda::negation_op::Negation>(const tensor_t*, tensor_t*, cuda::negation_op::Negation, hipStream_t);

// LOG
template void cuda::unaryOp<float, float, cuda::log_op::Log>(const tensor_t*, tensor_t*, cuda::log_op::Log, hipStream_t);
template void cuda::unaryOp<double, double, cuda::log_op::Log>(const tensor_t*, tensor_t*, cuda::log_op::Log, hipStream_t);

// EXP
template void cuda::unaryOp<float, float, cuda::exp_op::Exp>(const tensor_t*, tensor_t*, cuda::exp_op::Exp, hipStream_t);
template void cuda::unaryOp<double, double, cuda::exp_op::Exp>(const tensor_t*, tensor_t*, cuda::exp_op::Exp, hipStream_t);

// SQRT
template void cuda::unaryOp<float, float, cuda::sqrt_op::Sqrt>(const tensor_t*, tensor_t*, cuda::sqrt_op::Sqrt, hipStream_t);
template void cuda::unaryOp<double, double, cuda::sqrt_op::Sqrt>(const tensor_t*, tensor_t*, cuda::sqrt_op::Sqrt, hipStream_t);

// ABS
template void cuda::unaryOp<int8_t, int8_t, cuda::abs_op::Abs>(const tensor_t*, tensor_t*, cuda::abs_op::Abs, hipStream_t);
template void cuda::unaryOp<int16_t, int16_t, cuda::abs_op::Abs>(const tensor_t*, tensor_t*, cuda::abs_op::Abs, hipStream_t);
template void cuda::unaryOp<int32_t, int32_t, cuda::abs_op::Abs>(const tensor_t*, tensor_t*, cuda::abs_op::Abs, hipStream_t);
template void cuda::unaryOp<int64_t, int64_t, cuda::abs_op::Abs>(const tensor_t*, tensor_t*, cuda::abs_op::Abs, hipStream_t);
template void cuda::unaryOp<float, float, cuda::abs_op::Abs>(const tensor_t*, tensor_t*, cuda::abs_op::Abs, hipStream_t);
template void cuda::unaryOp<double, double, cuda::abs_op::Abs>(const tensor_t*, tensor_t*, cuda::abs_op::Abs, hipStream_t);

// SIN
template void cuda::unaryOp<float, float, cuda::sin_op::Sin>(const tensor_t*, tensor_t*, cuda::sin_op::Sin, hipStream_t);
template void cuda::unaryOp<double, double, cuda::sin_op::Sin>(const tensor_t*, tensor_t*, cuda::sin_op::Sin, hipStream_t);

// COS
template void cuda::unaryOp<float, float, cuda::cos_op::Cos>(const tensor_t*, tensor_t*, cuda::cos_op::Cos, hipStream_t);
template void cuda::unaryOp<double, double, cuda::cos_op::Cos>(const tensor_t*, tensor_t*, cuda::cos_op::Cos, hipStream_t);

// TAN
template void cuda::unaryOp<float, float, cuda::tan_op::Tan>(const tensor_t*, tensor_t*, cuda::tan_op::Tan, hipStream_t);
template void cuda::unaryOp<double, double, cuda::tan_op::Tan>(const tensor_t*, tensor_t*, cuda::tan_op::Tan, hipStream_t);

// SINH
template void cuda::unaryOp<float, float, cuda::sinh_op::Sinh>(const tensor_t*, tensor_t*, cuda::sinh_op::Sinh, hipStream_t);
template void cuda::unaryOp<double, double, cuda::sinh_op::Sinh>(const tensor_t*, tensor_t*, cuda::sinh_op::Sinh, hipStream_t);

// COSH
template void cuda::unaryOp<float, float, cuda::cosh_op::Cosh>(const tensor_t*, tensor_t*, cuda::cosh_op::Cosh, hipStream_t);
template void cuda::unaryOp<double, double, cuda::cosh_op::Cosh>(const tensor_t*, tensor_t*, cuda::cosh_op::Cosh, hipStream_t);

// TANH
template void cuda::unaryOp<float, float, cuda::tanh_op::Tanh>(const tensor_t*, tensor_t*, cuda::tanh_op::Tanh, hipStream_t);
template void cuda::unaryOp<double, double, cuda::tanh_op::Tanh>(const tensor_t*, tensor_t*, cuda::tanh_op::Tanh, hipStream_t);