#include "hip/hip_runtime.h"
#include <cstdint>   
#include <cstdint>
#include <array>
#include <stdexcept>
#include "cuda/exc.cuh"
#include "cuda/fns.cuh"  

namespace {
    
template<typename S, typename D, class Fn>
__global__ void scalarFnKernel(const S* src, D* dst, Fn fn) { 
    *dst = fn(*src);
}  

template<typename S, typename D, class Fn>
__global__ void batchedFnKernel(
    const S* src, shape_t src_shape, strides_t src_strides,
    D* dst, shape_t dst_shape, strides_t dst_strides,
    uint8_t rank, size_t ne, Fn fn
) { 
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < ne; idx += blockDim.x * gridDim.x) { 
        size_t offs = 0;
        size_t remaining = idx;

        for (int dim = rank - 1; dim >= 0; --dim) {
            size_t dim_idx = remaining % dst_shape.sizes[dim];
            remaining /= dst_shape.sizes[dim];
 
            size_t src_idx = (src_shape.sizes[dim] == 1) ? 0 : dim_idx;
            offs += src_idx * src_strides.sizes[dim];
        }

        dst[idx] = fn(src[offs]);
    }
}   

template<typename S, typename D, class Fn, class ... Args>
status launchFnKernel(const tensor_t* src, tensor_t* dst, stream_t stream, Args... args)  { 
    hipStream_t cudaStream = reinterpret_cast<hipStream_t>(stream.address);
    Fn fn(std::forward<Args>(args)...);
    if (src->rank == 0) {
        scalarFnKernel<S, D, Fn><<<1, 1, 0, cudaStream>>>(
            (const S*)(src->address),
            (D*)(dst->address), fn
        ); 
    } 
    
    else {
        size_t ne = 1;
        for (uint8_t dim = 0; dim < src->rank; ++dim) {
            ne *= dst->shape.sizes[dim];
        }

        size_t blockSize = 256;
        size_t gridSize = (ne + blockSize - 1) / blockSize;

        batchedFnKernel<S, D, Fn><<<gridSize, blockSize, 0, cudaStream>>>(
            (const S*)(src->address), src->shape, src->strides,
            (D*)(dst->address), dst->shape, dst->strides,
            src->rank, ne, fn
        ); 
    }  
    return SUCCESS;
} 
 
struct Log { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(log(a))) {
        return log(a);
    }
};
 
struct Exp { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(exp(a))) {
        return exp(a);
    }
};
  
struct Sqrt { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(sqrt(a))) {
        return sqrt(a);
    }
};

struct Rsqrt {
    float eps; 

    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept {
        if constexpr (std::is_same_v<std::decay_t<A>, float>) {
            return rsqrtf(a + eps);
        } else {
            return 1.0 / sqrt(a + eps);
        }
    }
}; 

struct Abs { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(abs(a))) {
        return abs(a);
    }
};
 
struct Sin { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(sin(a))) {
        return sin(a);
    }
};
 
struct Cos { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(cos(a))) {
        return cos(a);
    }
};

struct Tan { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(tan(a))) {
        return tan(a);
    }
}; 

struct Sinh { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(sinh(a))) {
        return sinh(a);
    }
};
 
struct Cosh{ 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(cosh(a))) {
        return cosh(a);
    }
};
 
struct Tanh { 
    template<class A>
    __device__ __forceinline__ auto operator()(A&& a) const noexcept(noexcept(tanh(a))) {
        return tanh(a);
    }
};    

} namespace cuda {

status log(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Log>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Log>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status exp(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Exp>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Exp>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status sqrt(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Sqrt>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Sqrt>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}


status rsqrt(const tensor_t* src, tensor_t* dst, stream_t stream, float eps) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Rsqrt>(src, dst, stream, eps);
        case float64:
            return launchFnKernel<double, double, Rsqrt>(src, dst, stream, eps);
        default:
            return UNSUPPORTED_DTYPE;
    }
}


status abs(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Abs>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Abs>(src, dst, stream);
        case int32:
            return launchFnKernel<int32_t, int32_t, Abs>(src, dst, stream);
        case int64:
            return launchFnKernel<int64_t, int64_t, Abs>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status sin(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Sin>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Sin>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status cos(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Cos>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Cos>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status tan(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Tan>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Tan>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status sinh(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Sinh>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Sinh>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status cosh(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Cosh>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Cosh>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status tanh(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float32:
            return launchFnKernel<float, float, Tanh>(src, dst, stream);
        case float64:
            return launchFnKernel<double, double, Tanh>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

} // namespace cuda
