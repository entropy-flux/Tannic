#include "hip/hip_runtime.h"
#include <cstdint>   
#include <cstdint>
#include <array>
#include <stdexcept>
#include <hip/hip_fp16.h> 
#include <thrust/complex.h>
#include "cuda/exc.cuh"
#include "cuda/ops.cuh"  

namespace {

template<typename S, typename D, class Op>
__global__ void singletonUnaryOpKernel(const S* __restrict__ src, D* __restrict__ dst, Op op) { 
    *dst = op(*src);
}  

template<typename S, typename D, class Op>
__global__ void contiguousUnaryOpKernel(const S* __restrict__ src, D* __restrict__ dst, size_t ne, Op op) {
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < ne; idx += blockDim.x * gridDim.x) {
        dst[idx] = op(src[idx]);
    }
}
 
template<typename S, typename D, class Op>
__global__ void stridedUnaryOpKernel(
    const S* __restrict__ src_ptr, strides_t src_strides,    
    D* __restrict__ dst_ptr, shape_t resets,          
    uint8_t dst_rank, size_t ne, Op op
){
    int rank = static_cast<int>(dst_rank);
    const size_t gstride = size_t(blockDim.x) * gridDim.x;
    for (size_t idx = size_t(blockIdx.x) * blockDim.x + threadIdx.x; idx < ne; idx += gstride) {
        size_t offset = 0;
        size_t divisor = 1;

        for (int dim = rank - 1; dim >= 0; --dim) { 
            const size_t extent    = resets.sizes[dim] / src_strides.sizes[dim];
            const size_t coord     = (idx / divisor) % extent; 
            offset += coord * src_strides.sizes[dim];
            divisor *= extent;
        }

        dst_ptr[idx] = op(src_ptr[offset]);
    }
} 
 
template<typename S, typename D, class Op, class ... Args>
status launchUnaryOpKernel(const tensor_t* src, tensor_t* dst, stream_t stream, Args... args)  { 
    hipStream_t cudaStream = reinterpret_cast<hipStream_t>(stream.address);
    Op op(std::forward<Args>(args)...);

    size_t ne = dst->size; 
    size_t blockSize = 256;
    size_t gridSize = (ne + blockSize - 1) / blockSize;

    switch (src->layout) {
        case SINGLETON: {
            singletonUnaryOpKernel<S, D, Op><<<1, 1, 0, cudaStream>>>(
                (const S*)(src->address),
                (D*)(dst->address),
                op
            ); 
            return SUCCESS;
        }

        case CONTIGUOUS: {
            contiguousUnaryOpKernel<S, D, Op><<<gridSize, blockSize, 0, cudaStream>>>(
                (const S*)(src->address),
                (D*)(dst->address),
                ne,
                op
            );
            return SUCCESS;
        }

        case STRIDED: {  
            strides_t strides{0};
            shape_t resets{0};
            for (int dim = 0; dim < src->rank; ++dim) {
                resets.sizes[dim] = dst->shape.sizes[dim] * src->strides.sizes[dim];
                strides.sizes[dim] = src->strides.sizes[dim];
            } 
            
            stridedUnaryOpKernel<S, D, Op><<<gridSize, blockSize, 0, cudaStream>>>(
                (const S*)(src->address), strides,
                (D*)(dst->address), resets,
                src->rank, ne,
                op
            );
            return SUCCESS;
        }

        default:
            return ERROR;
    } 
}   

#include <hip/hip_fp16.h>
#include <type_traits>

struct Neg { 
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return -a;
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __hneg(a);  
    } 
};

struct Cpy { 
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const noexcept {
        return a;
    }
};

struct Log { 
    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(logf(__half2float(a)));
    }
     
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return log(a);
    }
};

struct Exp { 
    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(expf(__half2float(a)));
    }
     
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return exp(a);
    }
};

struct Sqrt { 
    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(sqrtf(__half2float(a)));
    }
     
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return sqrt(a);
    }
};

struct Rsqrt {
    float eps; 
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return 1.0 / sqrt(a + eps);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(rsqrtf(__half2float(a) + eps));
    }
     
    __device__ __forceinline__ float operator()(float a) const {
        return rsqrtf(a + eps);
    } 
};

struct Abs {
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return abs(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __habs(a);
    } 
};

struct Sin {
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return sin(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(sinf(__half2float(a)));
    } 
};

struct Cos {
    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return cos(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(cosf(__half2float(a)));
    } 
};

struct Tan {

    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return tan(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(tanf(__half2float(a)));
    } 
};

struct Sinh {

    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return sinh(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(sinhf(__half2float(a)));
    } 
};

struct Cosh {

    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return cosh(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(coshf(__half2float(a)));
    }
     
};

struct Tanh { 

    template<class A>
    __device__ __forceinline__ auto operator()(A a) const {
        return tanh(a);
    }

    __device__ __forceinline__ __half operator()(__half a) const {
        return __float2half(tanhf(__half2float(a)));
    } 
};

} namespace cuda {

status neg(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case int8:
            return launchUnaryOpKernel<int8_t, int8_t, Neg>(src, dst, stream);
        case int16:
            return launchUnaryOpKernel<int16_t, int16_t, Neg>(src, dst, stream);
        case int32:
            return launchUnaryOpKernel<int32_t, int32_t, Neg>(src, dst, stream);
        case int64:
            return launchUnaryOpKernel<int64_t, int64_t, Neg>(src, dst, stream);
        case float16:
            return launchUnaryOpKernel<__half, __half, Neg>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Neg>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Neg>(src, dst, stream);
        case complex64:
            return launchUnaryOpKernel<thrust::complex<float>, thrust::complex<float>, Neg>(src, dst, stream);
        case complex128:
            return launchUnaryOpKernel<thrust::complex<double>, thrust::complex<double>, Neg>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status cpy(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case int8:
            return launchUnaryOpKernel<int8_t, int8_t, Cpy>(src, dst, stream);
        case int16:
            return launchUnaryOpKernel<int16_t, int16_t, Cpy>(src, dst, stream); 
        case int32:
            return launchUnaryOpKernel<int32_t, int32_t, Cpy>(src, dst, stream); 
        case int64:
            return launchUnaryOpKernel<int64_t, int64_t, Cpy>(src, dst, stream); 
        case float16:
            return launchUnaryOpKernel<__half, __half, Cpy>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Cpy>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Cpy>(src, dst, stream);
        case complex64:
            return launchUnaryOpKernel<thrust::complex<float>, thrust::complex<float>, Cpy>(src, dst, stream);
        case complex128:
            return launchUnaryOpKernel<thrust::complex<double>, thrust::complex<double>, Cpy>(src, dst, stream);
        default: 
            return UNSUPPORTED_DTYPE;
    }
}

status log(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Log>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Log>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Log>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status exp(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Exp>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Exp>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Exp>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status sqrt(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Sqrt>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Sqrt>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Sqrt>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status rsqrt(const tensor_t* src, tensor_t* dst, stream_t stream, float eps) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Rsqrt>(src, dst, stream, eps);
        case float32:
            return launchUnaryOpKernel<float, float, Rsqrt>(src, dst, stream, eps);
        case float64:
            return launchUnaryOpKernel<double, double, Rsqrt>(src, dst, stream, eps);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status abs(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Abs>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Abs>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Abs>(src, dst, stream);
        case int32:
            return launchUnaryOpKernel<int32_t, int32_t, Abs>(src, dst, stream);
        case int64:
            return launchUnaryOpKernel<int64_t, int64_t, Abs>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status sin(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Sin>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Sin>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Sin>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status cos(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Cos>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Cos>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Cos>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status tan(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Tan>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Tan>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Tan>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status sinh(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Sinh>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Sinh>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Sinh>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status cosh(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Cosh>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Cosh>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Cosh>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
}

status tanh(const tensor_t* src, tensor_t* dst, stream_t stream) {
    switch (src->dtype) {
        case float16:
            return launchUnaryOpKernel<__half, __half, Tanh>(src, dst, stream);
        case float32:
            return launchUnaryOpKernel<float, float, Tanh>(src, dst, stream);
        case float64:
            return launchUnaryOpKernel<double, double, Tanh>(src, dst, stream);
        default:
            return UNSUPPORTED_DTYPE;
    }
} 

} // namespace cuda