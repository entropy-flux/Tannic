#include <hip/hip_runtime.h>
#include <iostream> 
#include <stdexcept>

namespace cuda {

void checkError(hipError_t err, const char* file, int line, const char* expr) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) in '%s'\n",
                file, line, err, hipGetErrorString(err), expr);
        exit(EXIT_FAILURE);
    }
}
 
} // namespace cuda