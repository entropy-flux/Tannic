#include "cuda/cuda.cuh"

void cuda::checkError(hipError_t status, const char* message, const char* file, int line) {
    if (status != hipSuccess) {
        std::ostringstream error;
        error << "CUDA Error at " << file << ":" << line << "\n"
              << "  Code: " << static_cast<int>(status) << " (" << hipGetErrorName(status) << ")\n"
              << "  Message: " << hipGetErrorString(status);
        if (message && *message)
            error << "\n  Context: " << message;
        throw std::runtime_error(error.str());
    }
}