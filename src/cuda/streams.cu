#include "cuda/streams.cuh" 
#include <vector>
#include <forward_list>
#include <hip/hip_runtime.h> 
#include "cuda/exc.cuh"

namespace cuda {   
    
Streams::Streams() {
    int count;
    hipError_t err = hipGetDeviceCount(&count);  
    streams_.resize(count);
}

Streams::~Streams() {
    for (auto& device : streams_) {
        for (hipStream_t stream : device) {
            hipStreamDestroy(stream); 
        }
    }
}

hipStream_t Streams::pop(int device) {
    auto& streams = streams_[device];
    if (streams.empty()) {
        hipStream_t stream;
        hipStreamCreate(&stream);
        return stream;
    } else {
        hipStream_t stream = streams.front();
        streams.pop_front();
        return stream;
    }
}

void Streams::put(int device, hipStream_t stream) {
    streams_[device].push_front(stream);
}

}