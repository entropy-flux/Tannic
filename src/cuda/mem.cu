#include "cuda/exc.cuh"
#include "cuda/mem.cuh"
#include "cuda/streams.cuh"

namespace cuda { 
  
int getDeviceCount() {
    int count;
    hipError_t err = hipGetDeviceCount(&count); CUDA_CHECK(err);
    return count;
}

void setDevice(int id) {
    CUDA_CHECK(hipSetDevice(id));
}

void* allocate(const device_t* resource, size_t nbytes) {
    setDevice(resource->id); 
    void* ptr = nullptr;
    if (resource->traits & SYNC) { 
        CUDA_CHECK(hipMalloc(&ptr, nbytes));
    } else {
        Streams& streams = Streams::instance();
        hipStream_t stream = streams.pop(resource->id);
        CUDA_CHECK(hipMallocAsync(&ptr, nbytes, stream));
        streams.put(resource->id, stream);
    }
    return ptr;
} 

void* deallocate(const device_t* resource, void* ptr) {
    setDevice(resource->id);
    if (resource->traits & SYNC) {
        CUDA_CHECK(hipFree(ptr));
    } else {
        Streams& streams = Streams::instance();
        hipStream_t stream = streams.pop(resource->id);
        CUDA_CHECK(hipFreeAsync(ptr, stream));
        streams.put(resource->id, stream);
    }
    return nullptr;
}

void copyFromHost(const device_t* resource, const void* src , void* dst, size_t nbytes) {
    setDevice(resource->id);
    if (resource->traits & SYNC) {
        hipMemcpy(dst, src, nbytes, hipMemcpyHostToDevice);
    } 
    else {
        Streams& streams = Streams::instance();
        hipStream_t stream = streams.pop(resource->id);
        hipMemcpyAsync(dst, src, nbytes, hipMemcpyHostToDevice, stream);
        streams.put(resource->id, stream); 
    }
} 

bool compareFromHost(const device_t* resource, const void* hst_ptr, const void* dvc_ptr, size_t nbytes) {  
    void* buffer = malloc(nbytes); 
    CUDA_CHECK(hipMemcpy(buffer, dvc_ptr, nbytes, hipMemcpyDeviceToHost));
    bool result = (memcmp(hst_ptr, buffer, nbytes) == 0);
    free(buffer);   
    return result;
}

} // namespace cuda  