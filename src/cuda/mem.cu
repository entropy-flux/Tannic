#include <stdexcept>
#include "cuda/mem.cuh"
#include "cuda/exc.cuh"

namespace cuda {   

int getDeviceCount() {
    int count;
    hipError_t err = hipGetDeviceCount(&count); CUDA_CHECK(err);
    return count;
}

void* allocate(host_t const* resource, size_t nbytes) { 
    hipError_t err;  
    if (resource->traits & PINNED) {
        void* ptr = nullptr; 
        err = hipHostAlloc(&ptr, nbytes, hipHostMallocDefault); CUDA_CHECK(err);
        return ptr;
    }
    else {
        throw std::runtime_error("Allocation not supported by CUDA allocator.");
    }      
}

void deallocate(host_t const* resource, void* address, size_t nbytes) {  
    hipError_t err;   
    if (resource->traits & PINNED) {
        err = hipHostFree(address); CUDA_CHECK(err);
    }

    else {
        throw std::runtime_error("Dellocation not supported by CUDA allocator.");
    }
 
}

void* allocate(device_t const* resource, size_t nbytes) { 
    hipError_t err;  
    void* ptr = nullptr;
    err = hipSetDevice(resource->id);  CUDA_CHECK(err);
    err = hipMalloc(&ptr, nbytes); CUDA_CHECK(err); 
    return ptr; 
}

void deallocate(device_t const* resource, void* address, size_t nbytes) {  
    hipError_t err;  
    err = hipSetDevice(resource->id); CUDA_CHECK(err);
    err = hipFree(address); CUDA_CHECK(err); 
}

} // namespace cuda 