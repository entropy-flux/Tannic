#include "cuda/cuda.cuh"

namespace cuda {   

void* allocate(allocator_t const* allocator, size_t nbytes) { 
    hipError_t err; 
    if (allocator->environment == HOST) {
        void* ptr = nullptr; 
        err = hipHostAlloc(&ptr, nbytes, hipHostMallocDefault); CUDA_CHECK(err);
        return ptr;
    } else {
        void* ptr = nullptr;
        err = hipSetDevice(allocator->resource.device.id);  CUDA_CHECK(err);
        err = hipMalloc(&ptr, nbytes); CUDA_CHECK(err); 
        return ptr;
    }
}

void deallocate(allocator_t const* allocator, void* address, size_t nbytes) {  
    if (allocator->environment == HOST) {
        hipHostFree(address);   
    } else {
        hipSetDevice(allocator->resource.device.id);
        hipFree(address);
    }
}

} // namespace cuda 