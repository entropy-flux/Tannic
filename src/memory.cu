#include "Memory/Resources.hpp"
#include <iostream>

Resources::Resources() {
    try {
        int count;    
        hipGetDeviceCount(&count);
        for(int id = 0; id < count; id++) {
            devices_.emplace_back(id);
        }
    } 
    
    catch (const std::exception& exception) {
        std::cerr << "Error: " << exception.what() << std::endl; 
    }
}
 
void* Device::allocate(std::size_t memory) {
    void* address = nullptr;
    hipError_t error = hipSetDevice(id);
    if (error != hipSuccess) {
        std::cerr << "hipSetDevice failed for device " << id << ": " << hipGetErrorString(error) << "\n";
        return nullptr;
    }
    error = hipMalloc(&address, memory);
    if (error != hipSuccess) {
        std::cerr << "hipMalloc failed for device " << id << ": " << hipGetErrorString(error) << "\n";
        return nullptr;
    }
    return address;
}

void Device::deallocate(void* address, std::size_t) {
    hipSetDevice(id);
    hipFree(address);
}