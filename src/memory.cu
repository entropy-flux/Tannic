#include "Resources.hpp"
#include <iostream>
 
void* Device::allocate(std::size_t memory) {
    void* address = nullptr;
    hipError_t error = hipSetDevice(id);
    if (error != hipSuccess) {
        std::cerr << "hipSetDevice failed for device " << id << ": " << hipGetErrorString(error) << "\n";
        return nullptr;
    }
    error = hipMalloc(&address, memory);
    if (error != hipSuccess) {
        std::cerr << "hipMalloc failed for device " << id << ": " << hipGetErrorString(error) << "\n";
        return nullptr;
    }
    return address;
}

void Device::deallocate(void* address, std::size_t) {
    hipSetDevice(id);
    hipFree(address);
}

void Device::copy(void* address, void const* value, std::size_t size, Processor processor) const {
    hipMemcpy(address, value, size, hipMemcpyHostToDevice);
}

bool Device::compare(void const* address, void const* value, std::size_t size, Processor processor) const {
    void* buffer = std::malloc(size);
    if (!buffer) {
        throw std::bad_alloc();
    }

    hipError_t status = hipMemcpy(buffer, address, size, hipMemcpyDeviceToHost);
    if (status != hipSuccess) {
        std::free(buffer);
        throw std::runtime_error("hipMemcpy failed during comparision: " + std::string(hipGetErrorString(status)));
    }

    bool result = std::memcmp(buffer, value, size) == 0;
    std::free(buffer);
    return result;
}

Resources::Resources() {
    try {
        int count;    
        hipGetDeviceCount(&count);
        for(int id = 0; id < count; id++) {
            devices_.emplace_back(id);
        }
    } 
    
    catch (const std::exception& exception) {
        std::cerr << "Error: " << exception.what() << std::endl; 
    }
}