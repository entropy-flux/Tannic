#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}


int main() {
    const int N = 1 << 16; // 65536
    size_t size = N * sizeof(float);

    float *h_A = (float *)malloc(size); // Host buffer.
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size); // Cuda buffer!.
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); 
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipGraph_t graph;
    hipGraphExec_t graphExec;
 

    // Begin capturing
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    vectorAdd<<<(N + 255) / 256, 256, 0, stream>>>(d_A, d_B, d_C, N);
    hipStreamEndCapture(stream, &graph);

    // Instantiate the graph
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);

    // Launch the graph
    hipGraphLaunch(graphExec, stream);
    hipStreamSynchronize(stream);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Validate
    for (int i = 0; i < 10; ++i)
        printf("C[%d] = %f\n", i, h_C[i]);

    // Clean up
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graphExec);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipStreamDestroy(stream);

    return 0;
}
