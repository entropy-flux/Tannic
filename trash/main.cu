#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

#define N 512

#define CHECK_CUDA(call)                                                   \
    do {                                                                  \
        hipError_t err = call;                                           \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__,       \
                    __LINE__, hipGetErrorString(err));                   \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    } while (0)

#define CHECK_CUBLAS(call)                                                \
    do {                                                                 \
        hipblasStatus_t status = call;                                    \
        if (status != HIPBLAS_STATUS_SUCCESS) {                           \
            fprintf(stderr, "cuBLAS error at %s:%d: code %d\n", __FILE__,\
                    __LINE__, status);                                    \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    } while (0)

int main() {
    // Select GPU 0 explicitly
    CHECK_CUDA(hipSetDevice(0));

    size_t size = N * N * sizeof(float);
    float alpha = 1.0f, beta = 0.0f;

    // Host allocations
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Device allocations
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, size));
    CHECK_CUDA(hipMalloc(&d_B, size));
    CHECK_CUDA(hipMalloc(&d_C, size));

    CHECK_CUDA(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    CHECK_CUBLAS(hipblasSgemm(handle,
                             HIPBLAS_OP_N, HIPBLAS_OP_N,
                             N, N, N,
                             &alpha,
                             d_A, N,
                             d_B, N,
                             &beta,
                             d_C, N));

    CHECK_CUDA(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));

    // Print first 5 results
    for (int i = 0; i < 5; ++i) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}