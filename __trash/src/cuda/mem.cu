#include <sstream>
#include <string>
#include <stdexcept>
#include <cstdio>
#include <cstdlib>
#include "cuda/cuda.cuh"

#include <sstream>
#include <string>
#include <stdexcept>
#include <cstdio>
#include <cstdlib>
#include "cuda/cuda.cuh"
#include "core/resources.h"

void* cuda::syncHostAllocate(std::size_t nbytes) { 
    void* address = nullptr;
    CUDA_CHECK(hipHostAlloc(&address,nbytes, hipHostMallocDefault));
    return address;
}

void cuda::syncHostDeallocate(void* address) {
    if (address != nullptr) { 
        CUDA_CHECK(hipHostFree(address));
    }
}

void* cuda::syncAllocate(std::size_t size, int device) { 
    void* address = nullptr;
    CUDA_CHECK(hipSetDevice(device));
    CUDA_CHECK(hipMalloc(&address, size)); 
    return address;
}

void cuda::syncDeallocate(void* address, int device) {
    if (address != nullptr) { 
        CUDA_CHECK(hipSetDevice(device));
        CUDA_CHECK(hipFree(address));
    }
}