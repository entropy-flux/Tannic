#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>   
#include <vector>
#include <forward_list>

enum type { 
    none,
    int8,
    int16,
    int32,
    int64,
    float32,
    float64,
    complex64,   
    complex128,  
    TYPES
};

enum environment {
    HOST,
    DEVICE
};

enum host {
    PAGEABLE = 1 << 0, 
    PINNED   = 1 << 1, 
    MAPPED   = 1 << 2      
};
 
 
struct host_t {
    enum host traits;
};

struct device_t {
    int id; 
}; 

struct allocator_t { 
    enum environment environment;
    union {
        struct host_t host;
        struct device_t device;
    } resource;
}; 

struct tensor_t {
    uint8_t rank;
    void* address;
    const uint32_t* shape;
    const int64_t* strides;  
    enum type dtype; 
};  
 
struct stream_t { 
    uintptr_t address;
};  
 

class Streams {
private:
    Streams() {
        int count;
        hipError_t err = hipGetDeviceCount(&count);  
        streams_.resize(count);
    }

    std::vector<std::forward_list<hipStream_t>> streams_;

public: 
    ~Streams() {
        for (auto& device : streams_) {
            for (hipStream_t stream : device) {
                hipStreamDestroy(stream); 
            }
        }
    }

    Streams(const Streams&) = delete;
    Streams& operator=(const Streams&) = delete;
    Streams(Streams&&) = delete;
    Streams& operator=(Streams&&) = delete;

    hipStream_t pop(int device) {
        auto& streams = streams_[device];
        if (streams.empty()) {
            hipStream_t stream;
            hipStreamCreate(&stream);
            return stream;
        } else {
            hipStream_t stream = streams.front();
            streams.pop_front();
            return stream;
        }
    }

    void put(int device, hipStream_t stream) {
        streams_[device].push_front(stream);
    }

    static Streams& instance() {
        static Streams instance;
        return instance;
    }
};       

stream_t pop_stream(const device_t* device) {
    stream_t s;
    hipStream_t stream = Streams::instance().pop(device->id);
    s.address = reinterpret_cast<uintptr_t>(stream);
    return s;
}

void put_stream(const device_t* device, stream_t s) {
    hipStream_t stream = reinterpret_cast<hipStream_t>(s.address);
    Streams::instance().put(device->id, stream);
}

#include <unordered_map>

class Cache {
    struct Chunk {
        void* address;
        size_t nbytes;
    };

private:
    std::unordered_map<uintptr_t, Chunk> chunks_; 
    Cache() = default;

public:
    Cache(const Cache&) = delete;
    Cache& operator=(const Cache&) = delete;
    Cache(Cache&&) = delete;
    Cache& operator=(Cache&&) = delete;

    ~Cache() {
        for (auto& [key, chunk] : chunks_) { 
            hipStream_t cudaStream = reinterpret_cast<hipStream_t>(key);
            hipFreeAsync(chunk.address, cudaStream);
        }
        chunks_.clear();

        int devices = 0;
        hipGetDeviceCount(&devices);
        for (int dvc = 0; dvc < devices; ++dvc) {
            hipSetDevice(dvc);            
            hipDeviceSynchronize();    
        }
    }

    static Cache& instance() {
        static Cache instance;
        return instance;
    }

    void* get(const stream_t* stream, size_t nbytes) {
        auto iterator = chunks_.find(stream->address);
        if (iterator == chunks_.end()) {
            hipStream_t cudaStream = reinterpret_cast<hipStream_t>(stream->address); 
            void* address = nullptr; 
            hipMallocAsync(&address, nbytes, cudaStream);
            chunks_.emplace(stream->address, Chunk{address, nbytes});
            return address;
        } 

        else {
            Chunk& chunk = iterator->second;
            if (nbytes > chunk.nbytes) {
                hipStream_t cudaStream = reinterpret_cast<hipStream_t>(iterator->first);
                hipFreeAsync(chunk.address, cudaStream);
                hipMallocAsync(&chunk.address, nbytes, cudaStream);
            }
            return chunk.address;
        }
    }
}; 

size_t lsizeof(uint8_t rank) {
    return rank * (sizeof(uint32_t) + sizeof(int64_t));
}



int main() { 
    device_t device{0};
    stream_t stream = pop_stream(&device);


    put_stream(&device, stream);
}